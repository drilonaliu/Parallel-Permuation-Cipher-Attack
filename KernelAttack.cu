#include "hip/hip_runtime.h"
//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include "KernelAttack.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void cudaAttack(char* text, char* encrypted, char* permutatedText1, int* permutations, int permutationLength, int textLength) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int permutation[8];
	int m = i * permutationLength;
	int factorial = f(permutationLength);
	if (i < factorial) {
		char pT[8];
		//Fill the permutation array
		for (int k = 0; k < permutationLength; k++) {
			permutation[k] = permutations[m + k];
		}
		bool found = true;
		for (int c = 0; c < textLength; c++) {
			int p = permutation[c % permutationLength];
			int j = (c / permutationLength) * permutationLength + p;
			pT[j] = text[c];

			//Dont bother going through all the text if one character is not same as the plain text.
			if (!(pT[j] == encrypted[j])) {
				found = false;
				break;
			}
		}

		if (found) {
			printf("\n\nAttack is successful! Permutation is: ");
			for (int m = 0; m < permutationLength; m++) {
				printf("%d", permutation[m]);
			}
		}
	}
}

__device__ int f(int n) {
	int f = 1;
	for (int i = 1; i <= n; i++) {
		f = f * i;
	}
	return f;
}