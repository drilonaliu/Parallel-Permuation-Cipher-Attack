#include "hip/hip_runtime.h"
#include "PermutationAttack.cuh"
#include "KernelAttack.cuh"

using namespace std;

void parallelAttackPermutationCipher(string text, string encryptedText) {

	//Original Text to char array
	char* text_arr = new char[text.length() + 1];
	strcpy(text_arr, text.c_str());
	int size = (text.length() + 1) * sizeof(char);

	//EncryptedText to charr array
	char* encrypted_arr = new char[encryptedText.length() + 1];
	strcpy(encrypted_arr, encryptedText.c_str());

	//Permutated Text
	char* permutated_arr = new char[text.length() + 1];

	//Device pointers
	char* d_text;
	char* d_encryptedText;
	char* d_permutatedText;
	int* d_permutations = 0;

	//Memory allocation
	hipMalloc((void**)&d_text, size);
	hipMalloc((void**)&d_encryptedText, size);
	hipMalloc((void**)&d_permutatedText, size);

	//Memory copy
	hipMemcpy(d_text, text_arr, size, hipMemcpyHostToDevice);
	hipMemcpy(d_encryptedText, encrypted_arr, size, hipMemcpyHostToDevice);

	for (int permutationLength = 8; permutationLength <= 8; permutationLength++) {
		//Get all permutations
		int* permutations = getAllPermutations(permutationLength);

		//Allocate memory for permutations
		hipMalloc((void**)&d_permutations, permutationLength * factorial(permutationLength) * sizeof(int));

		//Copy all 
		hipMemcpy(d_permutations, permutations, permutationLength * factorial(permutationLength) * sizeof(int), hipMemcpyHostToDevice);

		//Launch Kernel 
		int threads = 1024;
		int blocksPerThread = (int)ceil(factorial(permutationLength) *1.0/ 1024);
		cudaAttack << <blocksPerThread, threads>> > (d_text, d_encryptedText, d_permutatedText, d_permutations, permutationLength, text.length());

		//Wait for cuda to launch the next kernel
		hipDeviceSynchronize();
	}
}


int factorial(int n) {
	int f = 1;
	for (int i = 1; i <= n; i++) {
		f = f * i;
	}
	return f;
}


int* getAllPermutations(int permutationLength) {
	int* allPermutations = new int[permutationLength * factorial(permutationLength)];
	vector<int> elements(permutationLength);
	for (int i = 0; i < permutationLength; i++) {
		elements[i] = i;
	}

	int counter = 0;
	do {
		for (int i = 0; i < permutationLength; i++) {
			allPermutations[permutationLength * counter + i] = elements[i];
		}
		counter += 1;
	} while (next_permutation(elements.begin(), elements.end()));

	return allPermutations;
}